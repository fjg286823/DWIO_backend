#include "hip/hip_runtime.h"
#include "include/common.h"
#include "../cuda/include/ITMSceneReconstructionEngine_CUDA.h"
#include "../cuda/include/ITMSceneReconstructionEngineShared.h"
#include "../hash/ITMRepresentationAccess.h"

using Vec3ida = Eigen::Matrix<int, 3, 1, Eigen::DontAlign>;

namespace DWIO
{
    namespace internal
    {
        namespace cuda
        {

            __device__ float get_tsdf(const ITMVoxel_d *localVBA, const ITMHashEntry *hashTable, float grid0, float grid1, float grid2)
            {
                Vector3i globalPos;
                globalPos << __float2int_rd(grid0), __float2int_rd(grid1), __float2int_rd(grid2);
                int vmIndex = 0;
                ITMVoxel_d voxel_get = readVoxel(localVBA, hashTable, globalPos, vmIndex);
                float tsdf_get = ITMVoxel_d::valueToFloat(voxel_get.tsdf);
                return tsdf_get;
            }

            __device__ float get_tsdf(const ITMVoxel_d *localVBA, const ITMHashEntry *hashTable, int grid0, int grid1, int grid2)
            {
                Vector3i globalPos;
                globalPos << grid0, grid1, grid2;
                int vmIndex = 0;
                ITMVoxel_d voxel_get = readVoxel(localVBA, hashTable, globalPos, vmIndex);
                float tsdf_get = ITMVoxel_d::valueToFloat(voxel_get.tsdf);
                return tsdf_get;
            }

            __device__ __inline__ float interpolate_trilinearly(const Vec3fda &point, const ITMVoxel_d *localVBA, const ITMHashEntry *hashTable)
            {
                Vec3ida point_in_grid = point.cast<int>();

                const float vx = (static_cast<float>(point_in_grid.x()) + 0.5f);
                const float vy = (static_cast<float>(point_in_grid.y()) + 0.5f);
                const float vz = (static_cast<float>(point_in_grid.z()) + 0.5f);

                point_in_grid.x() = (point.x() < vx) ? (point_in_grid.x() - 1) : point_in_grid.x();
                point_in_grid.y() = (point.y() < vy) ? (point_in_grid.y() - 1) : point_in_grid.y();
                point_in_grid.z() = (point.z() < vz) ? (point_in_grid.z() - 1) : point_in_grid.z();

                const float a = (point.x() - (static_cast<float>(point_in_grid.x()) + 0.5f));
                const float b = (point.y() - (static_cast<float>(point_in_grid.y()) + 0.5f));
                const float c = (point.z() - (static_cast<float>(point_in_grid.z()) + 0.5f));

                float tsdf_xyz = get_tsdf(localVBA, hashTable, point_in_grid.x(), point_in_grid.y(), point_in_grid.z());
                float tsdf_xyz1 = get_tsdf(localVBA, hashTable, point_in_grid.x(), point_in_grid.y(), point_in_grid.z() + 1);
                float tsdf_xy1z = get_tsdf(localVBA, hashTable, point_in_grid.x(), point_in_grid.y() + 1, point_in_grid.z());
                float tsdf_xy1z1 = get_tsdf(localVBA, hashTable, point_in_grid.x(), point_in_grid.y() + 1, point_in_grid.z() + 1);
                float tsdf_x1yz = get_tsdf(localVBA, hashTable, point_in_grid.x() + 1, point_in_grid.y(), point_in_grid.z());
                float tsdf_x1yz1 = get_tsdf(localVBA, hashTable, point_in_grid.x() + 1, point_in_grid.y(), point_in_grid.z() + 1);
                float tsdf_x1y1z = get_tsdf(localVBA, hashTable, point_in_grid.x() + 1, point_in_grid.y() + 1, point_in_grid.z());
                float tsdf_x1y1z1 = get_tsdf(localVBA, hashTable, point_in_grid.x() + 1, point_in_grid.y() + 1, point_in_grid.z() + 1);

                return tsdf_xyz * (1 - a) * (1 - b) * (1 - c) +
                       tsdf_xyz1 * (1 - a) * (1 - b) * c +
                       tsdf_xy1z * (1 - a) * b * (1 - c) +
                       tsdf_xy1z1 * (1 - a) * b * c +
                       tsdf_x1yz * a * (1 - b) * (1 - c) +
                       tsdf_x1yz1 * a * (1 - b) * c +
                       tsdf_x1y1z * a * b * (1 - c) +
                       tsdf_x1y1z1 * a * b * c;
            }

            __global__ void raycast_tsdf_kernel(ITMVoxel_d *localVBA,
                                                const ITMHashEntry *hashTable,
                                                PtrStepSz<uchar3> shading_buffer,
                                                const float voxel_scale,
                                                const float truncation_distance,
                                                const CameraConfiguration cam_parameters,
                                                const float3 init_pos,
                                                const Eigen::Matrix<float, 3, 3, Eigen::DontAlign> rotation,
                                                const Vec3fda translation)
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= shading_buffer.cols || y >= shading_buffer.rows)
                {
                    return;
                }

                const Vec3fda pixel_position((x - cam_parameters.principal_x) / cam_parameters.focal_x,
                                             (y - cam_parameters.principal_y) / cam_parameters.focal_y,
                                             1.f);

                Vec3fda ray_direction = rotation * pixel_position;
                ray_direction.normalize();

                float ray_length = voxel_scale;

                Vec3fda grid = (translation + (ray_direction * ray_length)) / voxel_scale;

                float tsdf = get_tsdf(localVBA, hashTable, grid(0), grid(1), grid(2));
                float previous_tsdf;

                float result_x = 0;
                float result_y = 0;
                float result_z = 0;

                float normal_x = 0;
                float normal_y = 0;
                float normal_z = 0;

                float max_search_length = 9000; // 由于原始最大搜索长度为5166.66修改为9米

                while (ray_length < max_search_length)
                {
                    ray_length += truncation_distance * 0.5f;
                    grid = (translation + (ray_direction * (ray_length + truncation_distance * 0.5f))) / voxel_scale;

                    previous_tsdf = tsdf;

                    tsdf = get_tsdf(localVBA, hashTable, grid(0), grid(1), grid(2));

                    if (previous_tsdf < 0.f && tsdf > 0.f)
                    {
                        break;
                    }
                    if (previous_tsdf > 0.f && tsdf < 0.f)
                    {
                        const float t_star = ray_length - truncation_distance * 0.5f * tsdf / (tsdf - previous_tsdf);

                        const auto vertex = translation + ray_direction * t_star;

                        const Vec3fda location_in_grid = (vertex / voxel_scale);

                        Vec3fda normal, shifted;

                        shifted = location_in_grid;
                        shifted.x() += 1;
                        const float Fx1 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        shifted = location_in_grid;
                        shifted.x() -= 1;
                        const float Fx2 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        normal.x() = (Fx1 - Fx2);

                        shifted = location_in_grid;
                        shifted.y() += 1;
                        const float Fy1 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        shifted = location_in_grid;
                        shifted.y() -= 1;
                        const float Fy2 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        normal.y() = (Fy1 - Fy2);

                        shifted = location_in_grid;
                        shifted.z() += 1;
                        const float Fz1 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        shifted = location_in_grid;
                        shifted.z() -= 1;
                        const float Fz2 = interpolate_trilinearly(shifted, localVBA, hashTable);

                        normal.z() = (Fz1 - Fz2);

                        if (normal.norm() == 0)
                            break;

                        normal.normalize();
                        result_x = vertex.x();
                        result_y = vertex.y();
                        result_z = vertex.z();

                        normal_x = normal.x();
                        normal_y = normal.y();
                        normal_z = normal.z();

                        break;
                    }
                }

                if (result_x == 0 && result_y == 0 && result_z == 0)
                {
                    return;
                }

                if (normal_x == 0 && normal_y == 0 && normal_z == 0)
                {
                    return;
                }

                const float kd_x = 98. / 255;
                const float kd_y = 121. / 255;
                const float kd_z = 148. / 255;

                const float light_position_x = init_pos.x;
                const float light_position_y = init_pos.y;
                const float light_position_z = init_pos.z;

                const float eye_position_x = translation.x();
                const float eye_position_y = translation.y();
                const float eye_position_z = translation.z();
                const float light_intensity = 0.8;

                float eye_pose_direction_x = eye_position_x - result_x;
                float eye_pose_direction_y = eye_position_y - result_y;
                float eye_pose_direction_z = eye_position_z - result_z;
                float lens = sqrt(pow(eye_pose_direction_x, 2) +
                                  pow(eye_pose_direction_y, 2) +
                                  pow(eye_pose_direction_z, 2));
                eye_pose_direction_x /= lens;
                eye_pose_direction_y /= lens;
                eye_pose_direction_z /= lens;

                float light_direction_x = light_position_x - result_x;
                float light_direction_y = light_position_y - result_y;
                float light_direction_z = light_position_z - result_z;

                lens = sqrt(pow(light_direction_x, 2) +
                            pow(light_direction_y, 2) +
                            pow(light_direction_z, 2));
                light_direction_x /= lens;
                light_direction_y /= lens;
                light_direction_z /= lens;

                const float ambinent_light_x = 0.1;
                const float ambinent_light_y = 0.1;
                const float ambinent_light_z = 0.1;
                float light_cos = normal_x * light_direction_x + normal_y * light_direction_y + normal_z * light_direction_z;
                if (light_cos <= 0)
                {
                    light_cos = -light_cos;
                }

                float light_coffi = light_intensity * light_cos;
                float diffuse_light_x = kd_x * light_coffi;
                float diffuse_light_y = kd_y * light_coffi;
                float diffuse_light_z = kd_z * light_coffi;

                float h_x = light_direction_x + eye_pose_direction_x;
                float h_y = light_direction_y + eye_pose_direction_y;
                float h_z = light_direction_z + eye_pose_direction_z;
                lens = sqrt(pow(h_x, 2) + pow(h_y, 2) + pow(h_z, 2));
                h_x /= lens;
                h_y /= lens;
                h_z /= lens;
                float h_cos = normal_x * h_x + normal_y * h_y + normal_z * h_z;
                if (h_cos < 0)
                {
                    h_cos = -h_cos;
                }

                light_coffi = light_intensity * pow(h_cos, 10);

                float specular_light_x = 0.5f * light_coffi;
                float specular_light_y = 0.5f * light_coffi;
                float specular_light_z = 0.5f * light_coffi;
                shading_buffer.ptr(y)[x].x = (uchar)((ambinent_light_x + diffuse_light_x + specular_light_x) * 255);
                shading_buffer.ptr(y)[x].y = (uchar)((ambinent_light_y + diffuse_light_y + specular_light_y) * 255);
                shading_buffer.ptr(y)[x].z = (uchar)((ambinent_light_z + diffuse_light_z + specular_light_z) * 255);
                if (ambinent_light_x + diffuse_light_x + specular_light_x > 1)
                {
                    shading_buffer.ptr(y)[x].x = 255;
                }
                if (ambinent_light_y + diffuse_light_y + specular_light_y > 1)
                {
                    shading_buffer.ptr(y)[x].y = 255;
                }
                if (ambinent_light_z + diffuse_light_z + specular_light_z > 1)
                {
                    shading_buffer.ptr(y)[x].z = 255;
                }
            }

            void SurfacePrediction(ITMScene<ITMVoxel_d, ITMVoxelBlockHash> *scene,
                                   const float &voxel_scale,
                                   GpuMat &shading_buffer,
                                   const float truncation_distance,
                                   const CameraConfiguration &cam_parameters,
                                   const float3 init_pos,
                                   cv::Mat &shading_img,
                                   const Eigen::Matrix4d &pose)
            {
                ITMVoxel_d *localVBA = scene->localVBA.GetVoxelBlocks();
                ITMHashEntry *hashTable = scene->index.GetEntries();

                dim3 threads(16, 16);
                dim3 blocks((shading_buffer.cols + threads.x - 1) / threads.x,
                            (shading_buffer.rows + threads.y - 1) / threads.y);

                cv::Scalar value = cv::Scalar(0, 0, 0);
                shading_buffer.setTo(value);

                raycast_tsdf_kernel<<<blocks, threads>>>(localVBA, hashTable, shading_buffer, voxel_scale, truncation_distance, cam_parameters, init_pos,
                                                         pose.block(0, 0, 3, 3).cast<float>(),
                                                         pose.block(0, 3, 3, 1).cast<float>());

                hipError_t cudaStatus = hipGetLastError();
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "[Surface Prediction] CUDA error: %s\n", hipGetErrorString(cudaStatus));
                }

                shading_buffer.download(shading_img);
            }
        }
    }
}